/*
 * =====================================================================================
 *
 *       Filename:  square_number.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016年09月30日 13时33分00秒
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  george, 
 *   Organization:  
 *
 * =====================================================================================
 */

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void fun(int * d_out,int *d_in){
		int idx = threadIdx.x;
		int num = d_in[idx];
		d_out[idx] = num*num*num;
}
int main(){
		const int ARRAY_SIZE = 96;
		const size_t  ARRAY_BYTES = ARRAY_SIZE* sizeof(int);
    		int h_in[ARRAY_SIZE];
		int h_out[ARRAY_SIZE];

		for(int i=0;i<ARRAY_SIZE;i++){
			h_in[i] = i;
		}
		int *d_in;
		int *d_out;
		hipMalloc(&d_in,ARRAY_BYTES);
		hipMalloc((void **)&d_out,ARRAY_BYTES);
		
		hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);
		fun<<<1,ARRAY_SIZE>>>(d_out,d_in);
		hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);
	
		printf("Output:\n");
		for(int i=0;i<ARRAY_SIZE;i++){
			printf("%d ",h_out[i]);
		}
		printf("\n");

		hipFree(d_in);
		hipFree(d_out);
}
